#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 16

// Kernel optimizado con memoria compartida
__global__ void matmul_shared_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float sum = 0.0f;
    for (int t = 0; t < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        if (row < M && t * BLOCK_SIZE + threadIdx.x < N)
            As[threadIdx.y][threadIdx.x] = A[row * N + t * BLOCK_SIZE + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;
        if (col < K && t * BLOCK_SIZE + threadIdx.y < N)
            Bs[threadIdx.y][threadIdx.x] = B[(t * BLOCK_SIZE + threadIdx.y) * K + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();
        for (int i = 0; i < BLOCK_SIZE; ++i)
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        __syncthreads();
    }
    if (row < M && col < K)
        C[row * K + col] = sum;
}

inline void checkCuda(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

extern "C" {
    void cuda_matmul(const float* A, const float* B, float* C, int M, int N, int K) {
        std::cout << "[CUDA] Llamando a cuda_matmul (GPU)" << std::endl;
        size_t size_A = M * N * sizeof(float);
        size_t size_B = N * K * sizeof(float);
        size_t size_C = M * K * sizeof(float);
        float *d_A, *d_B, *d_C;
        checkCuda(hipMalloc(&d_A, size_A), "hipMalloc d_A");
        checkCuda(hipMalloc(&d_B, size_B), "hipMalloc d_B");
        checkCuda(hipMalloc(&d_C, size_C), "hipMalloc d_C");
        checkCuda(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice), "hipMemcpy A");
        checkCuda(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice), "hipMemcpy B");
        dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
        dim3 blocks((K + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul_shared_kernel<<<blocks, threads>>>(d_A, d_B, d_C, M, N, K);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");
        checkCuda(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost), "hipMemcpy C");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    // Ejemplo de nueva función CUDA: suma de matrices
    __global__ void matrix_add_kernel(const float* A, const float* B, float* C, int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            C[idx] = A[idx] + B[idx];
        }
    }

    void cuda_matrix_add(const float* A, const float* B, float* C, int size) {
        std::cout << "[CUDA] Llamando a cuda_matrix_add (GPU)" << std::endl;
        float *d_A, *d_B, *d_C;
        checkCuda(hipMalloc(&d_A, size * sizeof(float)), "hipMalloc d_A");
        checkCuda(hipMalloc(&d_B, size * sizeof(float)), "hipMalloc d_B");
        checkCuda(hipMalloc(&d_C, size * sizeof(float)), "hipMalloc d_C");
        checkCuda(hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy A");
        checkCuda(hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy B");
        int threads = 256;
        int blocks = (size + threads - 1) / threads;
        matrix_add_kernel<<<blocks, threads>>>(d_A, d_B, d_C, size);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");
        checkCuda(hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy C");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
}
